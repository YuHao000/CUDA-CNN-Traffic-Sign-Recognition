
#include <hip/hip_runtime.h>

__global__ 
void bgr2gray(float *in_img, float *gray_img, int h, int w, int channel)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;

	if(i<h && j<w)
	{
		// R -> 0.299, G -> 0.587, B -> 0.114
		gray_img[i*w+j] = 0.299*in_img[0*h*w+i*w+j] + 0.587*in_img[1*h*w+i*w+j] + 0.114*in_img[2*h*w+i*w+j];		
	}

}
__device__ int hist[256];
__global__
void histogram_equalization(float *in_img, int h, int w, int num_levels)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;


	if(i*w+j<num_levels)
		hist[i*w+j] = 0;

	 __syncthreads();

	 if(i<h && j<w)
	 {
	 	atomicAdd(&(hist[(int)in_img[i*w+j]]),1);
	 }
	 __syncthreads();

}
__global__ void calcCDF(float* in_img , float* out_img, int h, int w, int num_levels)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;

	//Sequential Implementation -- can be modified to parallel using partial scan algorithm
	__shared__ float cdf[256];
	if(i*w + j == 0)
	{
		cdf[0] = (float)hist[i*w+j]/(h*w);
		for( int k = 1; k < num_levels; k++)
		{	
			cdf[k] = cdf[k-1] + (float)hist[k]/(h*w);
		}
	}
	__syncthreads();
	if(i*w+j<num_levels)
	 	cdf[i*w+j] = (float)(cdf[i*w+j] * (num_levels-1));
	__syncthreads();
	if(i<h && j<w)
	 	out_img[i*w+j] = (int)cdf[(int)in_img[i*w+j]];
}