#include "hip/hip_runtime.h"
__global__ 
void bgr2gray(float *in_img, float *gray_img, int h, int w, int channel)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;

	if(i<h && j<w)
	{
		// R -> 0.299, G -> 0.587, B -> 0.114
		gray_img[i*w+j] = 0.299*in_img[0*h*w+i*w+j] + 0.587*in_img[1*h*w+i*w+j] + 0.114*in_img[2*h*w+i*w+j];			
	}

}

__global__
void histogram_equalization(float *in_img, int *hist, float *out_img, int h, int w, int num_levels)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;

	__shared__ int hist[num_levels];

	if(i<16 && j<16)
		hist[i*16+j] = 0;

	__syncthreads();

	if(i<h && j<w)
	{
		hist[in_img[i*w+j]] += 1;
	}
	__syncthreads();

	if(i<16 && j<16)
	{
		hist[i*16+j] = (float)hist[i*16+j]/(h*w) * (num_levels-1);
	}
	__syncthreads();

	if(i==0 && j==0)
	{
		for(int x=1; x<num_levels;x++)
			hist[i] += hist[i-1];
	}
	__syncthreads();

	if(i<h && j<w)
		out_img[i*w+j] = hist[in_img[i*w+j]];

}